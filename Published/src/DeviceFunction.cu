#include "hip/hip_runtime.h"
#include "DeviceFunction.cuh"

__device__ void copy(double *a, gradient b, int number)
{
    double result = 0;
    for (int i = 0; i < number; i++)
    {
        a[i] = b[i];
    }
}

__device__ gradient GetGradient(state state1, control control, final_state final_state, QCost Q, RCost R)
{

    Eigen::VectorXd Q_result;

    Eigen::VectorXd tempj = (state1 - final_state);

    Q_result = 2 * Q.lazyProduct(tempj);

    Eigen::VectorXd R_result;
    R_result = 2 * R.lazyProduct(control);

    Matrix<double, ControlShape + StateShape, 1> temp;

    temp.topLeftCorner(StateShape, 1) = Q_result;
    temp.bottomRightCorner(ControlShape, 1) = R_result;

    return temp;
}

__device__ Hessian GetHessian(QCost Q, RCost R)
{
    int totalSize = StateShape + ControlShape;
    MatrixXd M = MatrixXd::Zero(totalSize, totalSize);

    // Place Q in the top-left corner
    M.topLeftCorner(StateShape, StateShape) = Q;

    // Place R in the bottom-right corner
    M.bottomRightCorner(ControlShape, ControlShape) = R;
    return M * 2;
}

__device__ equality GetEquality(state state1, control control, state initial)
{
    equality temp;
    temp[0] = (state1[0] - initial[0] - cos(initial[2]) * control[0]) * T;
    temp[1] = (state1[1] - initial[1] - sin(initial[2]) * control[0]) * T;
    temp[2] = (state1[2] - initial[2] - control[1]) * T;

    return temp;
}

__device__ JB GetJB1(state state1, state initial)
{
    StateJB StateJB;
    StateJB << 1, 0, 0,
        0, 1, 0,
        0, 0, 1;
    ControlJB ControlJB;
    ControlJB << -cos(initial[2]), 0,
        -sin(initial[2]), 0,
        0, -1;
    JB JB;
    JB.topLeftCorner(StateShape, StateShape) = StateJB;
    JB.bottomRightCorner(StateShape, ControlShape) = ControlJB;

    return JB;
}

__device__ JB GetJB2(state state1, control control)
{
    StateJB StateJB;
    StateJB << -1, 0, control[0] * sin(state1[2]),
        0, -1, -control[0] * cos(state1[2]),
        0, 0, -1;
    ControlJB ControlJB;
    ControlJB << 0, 0,
        0, 0,
        0, 0;
    JB JB;
    JB.topLeftCorner(StateShape, StateShape) = StateJB;
    JB.bottomRightCorner(StateShape, ControlShape) = ControlJB;

    return JB;
}

__device__ Hessian PsedoInverse(Hessian hessian)
{
    Hessian temp;
    temp.setZero();
    for (int i = 0; i < StateShape + ControlShape; i++)
    {
        temp.row(i)[i] = 1 / hessian.row(i)[i];
    }
    return temp;
}

__device__ void mycopy(SharedMatrix *shared, temp temp1, temp temp2, temp temp3, temp temp4, int idx)
{
    int base = (idx - 1) * StateShape;

    for (int i = 0; i < StateShape; i++)
    {
        for (int j = 0; j < StateShape; j++)
        {
            MyatomicAdd(&shared->row(i + base)[j + base], temp1.row(i)[j]);

            MyatomicAdd(&shared->row(i + base)[j + base + StateShape], temp2.row(i)[j]);

            MyatomicAdd(&shared->row(i + base + StateShape)[j + base], temp3.row(i)[j]);

            MyatomicAdd(&shared->row(i + base + StateShape)[j + base + StateShape], temp4.row(i)[j]);
        }
    }
}

__device__ void mycopy2(SharedMatrix *shared, temp temp1, int idx)
{
    int base = (idx - 1) * StateShape;

    for (int i = 0; i < StateShape; i++)
    {
        for (int j = 0; j < StateShape; j++)
        {
            MyatomicAdd(&shared->row(i + base)[j + base], temp1.row(i)[j]);
        }
    }
}

__device__ void DebugCopy(SharedMatrix *shared, SharedMatrix *debug)
{

    for (int i = 0; i < horizon * StateShape; i++)
    {
        for (int j = 0; j < horizon * StateShape; j++)
        {
            debug->row(i)[j] = shared->row(i)[j];
        }
    }
}

__device__ void SecondPhaseCopy(FirstPhaseDual *FirstDual, double *d_x, int idx)
{
    for (int i = 0; i < StateShape; i++)
    {
        FirstDual->row(i)[0] = d_x[i + (idx - 1) * StateShape];
    }
}

__device__ double MyatomicAdd(double *address, double val)
{
    unsigned long long int *address_as_ull =
        (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ double empty(SharedMatrix *matrix)
{
    for (int i = 0; i < StateShape * horizon; i++)
    {
        for (int j = 0; j < StateShape * horizon; j++)
        {
            matrix->row(i)[j] = 0;
        }
    }
}

int dense_to_csr1(UpperDualCache h_A_dense, SparseUpperDualCache *sparse, int Nrows, int Ncols,
                  RowIndices *h_A_RowIndices, ColIndices *h_A_ColIndices)
{

    int nnz = 0; // Number of non-zero elements encountered
    double val;

    for (int i = 0; i < Nrows; ++i)
    {
        h_A_RowIndices->row(i)[0] = nnz; //
        for (int j = 0; j < Ncols; ++j)
        {
            val = h_A_dense.coeff(j * Nrows + i); //
            if (abs(val - 0.0) > 1e-10)
            {

                // printf("val = %f " , val);
                // printf("j = %d " , j);
                // printf("nnz = %d \n" , nnz);

                sparse->row(nnz)[0] = val;
                h_A_ColIndices->row(nnz)[0] = j; // Store column index
                nnz++;
            }
        }
    }

    return nnz;

    // Last entry in h_A_RowIndices should be nnz
    // printf("asdfasdfas nnz = %d \n" , nnz);
    // h_A_RowIndices->row(Nrows)[0] = nnz;
}

__device__ int dense_to_csr(UpperDualCache cacheA, UpperDualCache cacheB, UpperDualCache cacheC, SparseUpperDualCache *sparse, int Nrows, int Ncols,
                            RowIndices *h_A_RowIndices, ColIndices *h_A_ColIndices)
{
    Eigen::Matrix<double, StateShape, StateShape * 3> temp;
    // temp << cacheA , `cacheB , cacheC;

    temp.block<StateShape, StateShape>(0, 0) = cacheA;
    temp.block<StateShape, StateShape>(0, StateShape) = cacheB;
    temp.block<StateShape, StateShape>(0, StateShape * 2) = cacheC;

    // if (threadIdx.x == 1)
    // {
    //     for (int i = 0; i < StateShape; i++)
    //     {
    //         for (int j = 0; j < StateShape * 3; j++)
    //         {
    //             printf("%f ", temp.row(i)[j]);
    //         }
    //         printf("\n");
    //     }
    // }

    int nnz = 0;
    double val;

    for (int i = 0; i < Nrows; ++i)
    {
        h_A_RowIndices->row(i)[0] = nnz; //
        for (int j = 0; j < Ncols; ++j)
        {
            val = temp.coeff(j * Nrows + i); //
            if (abs(val - 0.0) > 1e-10)
            {

                // printf("val = %f " , val);
                // printf("j = %d " , j);
                // printf("nnz = %d \n" , nnz);

                sparse->row(nnz)[0] = val;
                h_A_ColIndices->row(nnz)[0] = j; // Store column index
                nnz++;
            }
        }
    }


    // Last entry in h_A_RowIndices should be nnz
    // printf("asdfasdfas nnz = %d \n" , nnz);
    h_A_RowIndices->row(Nrows)[0] = nnz;
    return nnz;

}


__device__ void SparsecopyToPointer(const SparseUpperDualCache& matrix, tinytype* ptr, int a, int n, int I) {
    std::memcpy(ptr + I, matrix.data() + a, n * sizeof(tinytype));
}


__device__ void RowIndicescopyToPointer(const RowIndices& matrix, int* ptr, int a, int n, int I) {
    // std::memcpy(ptr + I, matrix.data() + a, n * sizeof(tinytype));
    for(int i = 0 ; i < n ; i ++)
    {
        ptr[I + i] = matrix.coeff(i + a);
    }
}

__device__ void ColIndicescopyToPointer(const ColIndices& matrix, int* ptr, int a, int n, int I) {
    for(int i = 0 ; i < n ; i ++)
    {
        ptr[I + i] = matrix.coeff(i + a);
    }
}