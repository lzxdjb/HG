#include "hip/hip_runtime.h"
#include "head.cuh"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
void debug(TinyCache *solvergpu)
{
    for (int i = 0; i < horizon + 1; i++)
    {
        // std::cout << "state_vectors = " << solvergpu[i].state1 << std::endl;
        // std::cout << "control_vectors = " << solvergpu[i].control << std::endl;

        // std::cout << "gradient = " << solvergpu[i].gradient << std::endl;
        // std::cout << "Hessian = " << solvergpu[i].Hessian << std::endl;

        // std::cout << "equality = " << solvergpu[i].equality << std::endl;

        // std::cout << "JB1 = \n" << solvergpu[i].JB1 << std::endl;

        // std::cout << "JB2 = \n" << solvergpu[i].JB2 << std::endl;

        // std::cout << "LowerLeftDown1 = \n" << solvergpu[i].LowerLeftDown1 << std::endl;

        // std::cout << "LowerLeftDown2 = \n" << solvergpu[i].LowerLeftDown2 << std::endl;

        // std::cout << "debug = \n" << solvergpu[i].debug << std::endl;

        // std::cout << "shared = \n" << &dshared << std::endl;

        // std::cout << "final_state = " << solvergpu[i].final_state << std::endl;
        // std::cout << "initstate = " << solvergpu[i].initial_state << std::endl;

        // std::cout << "FinalMatrix = \n"
        //           << solvergpu[i].OriginalMatrix << std::endl;
        // std::cout << "FinalColumn = \n"
        //           << solvergpu[i].FinalColumn << std::endl;

        // std::cout << "L = \n" << solvergpu[i].L << std::endl;

        // std::cout << "varible1 = \n" << solvergpu[i].varible1 << std::endl;
        // std::cout << "varible2 = \n"
        //           << solvergpu[i].varible2 << std::endl;

        // std::cout << "FirstVarible = \n"<< solvergpu[i].FirstVarible << std::endl;

        // std::cout << "FirstDual = \n" << solvergpu[i].FirstDual << std::endl;

        //  std::cout << "soltuion temp = \n"<< solvergpu[i].solutionTemp << std::endl;

        // std::cout << "convergence = \n"<< solvergpu[i].convergence << std::endl;
        std::cout<<"cache1 \n= " <<solvergpu[i].cache1<<std::endl;
        std::cout<<"cache2 \n= " <<solvergpu[i].cache2<<std::endl;
        std::cout<<"cache3 \n= " <<solvergpu[i].cache3<<std::endl;
        // std::cout<<"nnz = "<<solvergpu[i].nnz<<std::endl;
        // std::cout<<"SparseCache = "<<solvergpu[i].SparseCache.transpose()<<std::endl;
        // std::cout<<"h_A_RowIndices = " <<solvergpu[i].h_A_RowIndices.transpose()<<std::endl;
        // std::cout<<"h_A_ColIndices = " <<solvergpu[i].h_A_ColIndices.transpose()<<std::endl;

        //  std::cout<<"\n";
    }
}

__global__ void sparse_represent(TinyCache *solver_gpu, int *index ,  double * sparseMatix , int * RowIndices , int * ColIndices)
{
    int idx = threadIdx.x;
    if(idx < horizon + 1 && idx > 0)
    {
        SparsecopyToPointer(solver_gpu[idx].SparseCache , sparseMatix , 0  ,index[idx] - index[idx - 1] ,  index[idx - 1]);
        

        if(idx == 1)
        {
            RowIndicescopyToPointer(solver_gpu[idx].h_A_RowIndices , RowIndices , 0 , StateShape + 1 , (idx- 1) * StateShape);
        }
        else{


            for(int i = 1 ; i < StateShape + 1 ; i ++)
            {
                solver_gpu[idx].h_A_RowIndices.row(i)[0] += index[idx - 1];
            }
             RowIndicescopyToPointer(solver_gpu[idx].h_A_RowIndices , RowIndices , 1 , StateShape , (idx- 1) * StateShape + 1);
        }

        if(idx < 3)
        {
            ColIndicescopyToPointer(solver_gpu[idx].h_A_ColIndices , ColIndices , 0 , index[idx] - index[idx - 1] , index[idx - 1]);
        }
        else{

            for(int i = 0 ; i < index[idx] - index[idx - 1] ; i ++)
            {
                solver_gpu[idx].h_A_ColIndices.row(i)[0] += StateShape * (idx - 2);
            }

            ColIndicescopyToPointer(solver_gpu[idx].h_A_ColIndices , ColIndices , 0 , index[idx] - index[idx - 1] , index[idx - 1]);
        }

    }
}

__global__ void solve_kernel(TinyCache *solver_gpu, double *bigDual, QCost Q, RCost R, state init_state, state final_state , int* d_index)
{

    int idx = threadIdx.x;
    // if (idx < horizon + 1 && idx > 0)
    {

        gradient Allgradient;
        Hessian Hessian;
        equality equality;

        JB JB1;
        JB JB2;


        JB LowerLeftDown1;
        JB LowerLeftDown2;

        LowerLeftDown1.setZero();
        LowerLeftDown2.setZero();

        Hessian = GetHessian(Q, R);

    
        equality = GetEquality(solver_gpu[idx + 1].state1, solver_gpu[idx + 1].control, solver_gpu[idx].state1);

        solver_gpu[idx + 1].equality = equality;
        

        // if (idx + 1 != horizon)
        {

            JB1 = GetJB1(solver_gpu[idx + 1].state1, solver_gpu[idx].state1);
            JB2 = GetJB2(solver_gpu[idx + 1].state1, solver_gpu[idx + 1].control);
            
        }

        // else
        // {
        // //     // printf("asdfasdf");
        //     JB1 = GetJB1(solver_gpu[idx + 1].state1, solver_gpu[idx].state1);
        //     JB2.setZero();
        // }

        Allgradient = GetGradient(solver_gpu[idx + 1].state1, solver_gpu[idx + 1].control, final_state, Q, R);

        solver_gpu[idx + 1].JB1 = JB1;
        solver_gpu[idx + 1].JB2 = JB2;
        solver_gpu[horizon].JB2.setZero();


        LowerLeftDown1 = solver_gpu[idx + 1].JB1.lazyProduct(PsedoInverse(Hessian));
        LowerLeftDown2 = solver_gpu[idx + 1].JB2.lazyProduct(PsedoInverse(Hessian));
        
        solver_gpu[idx + 1].LowerLeftDown2 = LowerLeftDown2;

        //// for debug
              solver_gpu[idx + 1].LowerLeftDown1 = LowerLeftDown1;
        ////


        if (idx == 0)
        {

 
            solver_gpu[idx + 1].cache1.topLeftCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown1.lazyProduct(solver_gpu[idx + 1].JB1.transpose());

            solver_gpu[idx + 1].cache2.topLeftCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown1.lazyProduct(solver_gpu[idx + 1].JB2.transpose());

            solver_gpu[idx + 2].cache1.topLeftCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown2.lazyProduct(solver_gpu[idx + 1].JB1.transpose());

            solver_gpu[idx + 2].cache2.topLeftCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown2.lazyProduct(solver_gpu[idx + 1].JB2.transpose());
        }
        
/// my new idea
    // A.block<3, 3>(0, 0) = A1;
    // A.block<3, 3>(0, 3) = A2;
    // A.block<3, 3>(0, 6) = A3;
    //     solver_gpu[idx + 1].bigcache.block<3, 3> = 

//////
        else if (idx < horizon - 1)
        {

            solver_gpu[idx + 1].cache2.topLeftCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown1.lazyProduct(solver_gpu[idx + 1].JB1.transpose());

            // printf("idx = %d " , idx);

           solver_gpu[idx + 1].cache3.topRightCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown1.lazyProduct(solver_gpu[idx + 1].JB2.transpose());

            solver_gpu[idx + 2].cache1.topLeftCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown2.lazyProduct(solver_gpu[idx + 1].JB1.transpose());

            solver_gpu[idx + 2].cache2.topRightCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown2.lazyProduct(solver_gpu[idx + 1].JB2.transpose());

        }
        else
        {
            solver_gpu[idx + 1].cache2.topRightCorner(StateShape , StateShape) += -solver_gpu[idx + 1].LowerLeftDown1.lazyProduct(solver_gpu[idx + 1].JB1.transpose());
          
    
        }
        // solver_gpu[idx].FirstVarible = -Allgradient;

        // __syncthreads();

        // state SolutionTemp;
        // if (idx == 1)
        // {
        //     SolutionTemp = LowerLeftDown1.lazyProduct(solver_gpu[idx].FirstVarible);
        // }
        // else
        // {
        //     // printf("asdfasdfas");
        //     SolutionTemp = LowerLeftDown1.lazyProduct(solver_gpu[idx].FirstVarible);

        //     SolutionTemp += solver_gpu[idx - 1].LowerLeftDown2.lazyProduct(solver_gpu[idx - 1].FirstVarible);
        // }

        // solver_gpu[idx].FirstDual = -equality - SolutionTemp;
       
        solver_gpu[idx + 1].gradient = Allgradient;

        solver_gpu[idx + 1].Hessian = Hessian;

        // solver_gpu[idx].nnz = dense_to_csr(solver_gpu[idx].cache1 , solver_gpu[idx].cache2 , solver_gpu[idx].cache3 ,&solver_gpu[idx].SparseCache , StateShape , StateShape * 3 , &solver_gpu[idx].h_A_RowIndices , &solver_gpu[idx].h_A_ColIndices);

        // d_index[idx] = solver_gpu[idx].nnz;

        // for (int i = 0; i < StateShape; i++)
        // {
        //     bigDual[(idx - 1) * StateShape + i] = solver_gpu[idx].FirstDual[i];
        // }
    }
}

__global__ void Second_solve_kernel(TinyCache *solver_gpu, double *d_x )
{
    int idx = threadIdx.x;
    double learning_rate = 1;
    // if (idx < horizon + 1 && idx > 0)
    {
        SecondPhaseCopy(&solver_gpu[idx].FirstDual, d_x, idx);

        convergence SolutionTemp;

        if (idx == horizon)
        {
            SolutionTemp = solver_gpu[idx].JB1.transpose().lazyProduct(solver_gpu[idx].FirstDual);
        }
        else if(idx > 0)
        {
            // printf("asdfasdfas");
            SolutionTemp = solver_gpu[idx].JB1.transpose().lazyProduct(solver_gpu[idx].FirstDual);

            SolutionTemp += solver_gpu[idx].JB2.transpose().lazyProduct(solver_gpu[idx + 1].FirstDual);
        }
        solver_gpu[idx].FirstVarible -= SolutionTemp;
        solver_gpu[idx].FirstVarible = PsedoInverse(solver_gpu[idx].Hessian).lazyProduct(solver_gpu[idx].FirstVarible);

        // solver_gpu[idx].convergence = SolutionTemp;

        solver_gpu[idx].state1 +=  solver_gpu[idx].FirstVarible.topLeftCorner(StateShape , 1) * learning_rate;

        solver_gpu[idx].control +=  solver_gpu[idx].FirstVarible.bottomRightCorner(ControlShape , 1) * learning_rate;


    }
}

void tiny_solve_cuda(TinyCache *cache,  tinytype *bigDual, QCost Q, RCost R, state init_state, state final_state){
    TinyCache *solver_gpu;
    double *d_dual; // FirstStageDual
    hipMalloc(&d_dual, StateShape * horizon * sizeof(double));

    ////

    checkCudaErrors(hipMalloc((void **)&solver_gpu, sizeof(TinyCache) * (horizon + 1) ));
    // printf("addr %d\n", solver_gpu);
    checkCudaErrors(hipMemcpy(solver_gpu, cache, sizeof(TinyCache) * (horizon + 1), hipMemcpyHostToDevice));

    hipsparseHandle_t handle;
    (hipsparseCreate(&handle));
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
 
    hipsolverSpHandle_t solver_handle;
    hipsolverSpCreate(&solver_handle);
    int singularity;

////////##########
////////########## Do not delete

    int h_index[horizon + 1] = {0};
    int *d_index;
    checkCudaErrors(hipMalloc((void**)&d_index, sizeof(int) * (horizon + 1)));
    hipMemcpy(d_index, &h_index, sizeof(int) * (horizon + 1) , hipMemcpyHostToDevice);

    double *d_sparsematrix;
    int *d_RowIndices;
    int *d_ColIndices;
    checkCudaErrors(hipMalloc((void**)&d_sparsematrix, sizeof(double) * (StateShape * StateShape * horizon * 3)));
    checkCudaErrors(hipMalloc((void**)&d_RowIndices, sizeof(int) *  (StateShape * horizon + 1)));
    checkCudaErrors(hipMalloc((void**)&d_ColIndices, sizeof(int) * (StateShape * StateShape * horizon * 3 )));
    
    double * d_my_solution;
    checkCudaErrors(hipMalloc((void**)&d_my_solution, sizeof(double) * (horizon * StateShape)));

//@@@@@@@@@@@
    int nnz ;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    bool flag = false;
    for (int i = 0; i < 1; i++)
    {

        solve_kernel<<<1, horizon>>>(solver_gpu, d_dual , Q, R, init_state, final_state , d_index);
        checkCudaErrors(hipDeviceSynchronize());

        thrust::inclusive_scan(thrust::device_pointer_cast(d_index), thrust::device_pointer_cast(d_index) + horizon + 1, thrust::device_pointer_cast(d_index)); // in-place scan

        // checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipMemcpy(&nnz, d_index + horizon, 4, hipMemcpyDeviceToHost));

////###########$$$$$$$$$$$$ TEST

        sparse_represent<<<1 , horizon + 1>>>(solver_gpu , d_index , d_sparsematrix , d_RowIndices , d_ColIndices);
     
////Test
        (hipsolverSpDcsrlsvqr(solver_handle, StateShape * horizon, nnz, descrA, d_sparsematrix, d_RowIndices, d_ColIndices, d_dual, 0.000001, 0, d_my_solution, &singularity));

        // double * my_solution = (double *)malloc(StateShape * horizon* sizeof(double));


////##########&&&&&&&&&&&&& EndTEST
        Second_solve_kernel<<<1, horizon + 1>>>(solver_gpu, d_my_solution );
        checkCudaErrors(hipDeviceSynchronize());
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time for sparse_represent: " << milliseconds * 1e-3 << " s" << std::endl;

    checkCudaErrors(hipMemcpy(cache, solver_gpu, sizeof(TinyCache) * (horizon + 1), hipMemcpyDeviceToHost));

    // std::cout<<"my answer"<<std::endl;
    debug(cache);

}


